#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>

#include "../include/kmeans.cuh"

/**
 * @brief Calculates the squared euclidean distance between two points
 *
 * @param x
 * @param y
 * @return __device__
 */
__device__ inline static
float euclidean_distance(const sample* x, const cluster* y) {
    return (x->x - y->x) * (x->x - y->x) + (x->y - y->y) * (x->y - y->y);
}

__global__
void cluster_points(
    sample* sx,
    cluster* cx,
    const uint32_t N,
    const uint32_t K
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {

        // set min_dist to the first cluster
        float min_dist = euclidean_distance(&sx[idx], &cx[0]);
        sx[idx].id = 0;

        for (uint32_t i = 1; i < K; i++) {
            float dist = euclidean_distance(&sx[idx], &cx[i]);
            if (dist < min_dist) {
                min_dist = dist;
                sx[idx].id = i;
            }
        }
    }
}

__global__
void recalc_clusters(cluster* cx, const uint32_t K) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < K) {
        cx[idx].x = cx[idx]._x / cx[idx]._c;
        cx[idx].y = cx[idx]._y / cx[idx]._c;
    }
}

__global__
void do_work(
    sample* sx,
    cluster* cx,
    const uint32_t K,
    const uint32_t N
) {
    int tdx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tdx;
    // shared memory for the clusters
    extern __shared__ cluster s_cx[];

    // Clear the current clusters auxilary values.
    if (idx < K) {
        cx[idx]._x = 0.0;
        cx[idx]._y = 0.0;
        cx[idx]._c = 0;
    }

    // 
    if (tdx < K) {
        s_cx[tdx]._x = 0.0;
        s_cx[tdx]._y = 0.0;
        s_cx[tdx]._c = 0;
    }
    // synchronize threads before starting
    __syncthreads();

    // This part of the algorithm used to be in the cluster_points function
    // in the sequential/omp version. Due to changes in the algorithm, as well
    // as the fact that the structures were changed, this was moved here.
    if (idx < N) {
        int c_id = sx[idx].id;
        atomicAdd(&s_cx[c_id]._x, sx[idx].x);
        atomicAdd(&s_cx[c_id]._y, sx[idx].y);
        atomicAdd(&s_cx[c_id]._c, 1);
    }

    __syncthreads();

    if (tdx < K) {
        atomicAdd(&cx[tdx]._x, s_cx[tdx]._x);
        atomicAdd(&cx[tdx]._y, s_cx[tdx]._y);
        atomicAdd(&cx[tdx]._c, s_cx[tdx]._c);
    }
}


/**
 * @brief
 *
 * @param samples
 * @param clusters
 * @param K Cluster count
 * @param N Sample count
 */
uint32_t k_means_cuda(sample* sx, cluster* cx, uint32_t N, uint32_t K) {
    // Allocate device memory for clusters and samples.
    uint32_t iter = 0;
    sample* d_sx;
    cluster* d_cx;
    hipMalloc((void**)&d_sx, N * sizeof(sample));
    hipMalloc((void**)&d_cx, K * sizeof(cluster));

    // Copy samples and clusters to device memory.
    hipMemcpy(d_sx, sx, N * sizeof(sample), hipMemcpyHostToDevice);
    hipMemcpy(d_cx, cx, K * sizeof(cluster), hipMemcpyHostToDevice);

    // take the above for loop and convert it to a do-while loop
    do {
        // Step 1c - Assign each sample to the nearest cluster using the euclidean distance.
        cluster_points << < ceil(N / 256), 256 >> > (d_sx, d_cx, N, K);
        hipDeviceSynchronize();

        // Step 2 - Calculate the centroid of each cluster. (also known as geometric center)
        do_work << < ceil(N + 255) / 256, 256, K * sizeof(cluster) >> > (d_sx, d_cx, K, N);
        hipDeviceSynchronize();

        // Step 3 - Assign each sample to the nearest cluster using the euclidean distance.
        recalc_clusters << <ceil(K / 10), 10 >> > (d_cx, K);
        hipDeviceSynchronize();
        iter++;
    } while (iter < MAX_ITER);

    //// Copy the clusters back to the host.
    hipMemcpy(cx, d_cx, K * sizeof(cluster), hipMemcpyDeviceToHost);

    // Free the device memory.
    hipFree(d_sx);
    hipFree(d_cx);

    return iter;
}
