#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "../include/kmeans.cuh"

void generate_samples(sample* sx, cluster* cx, const uint32_t N, const uint32_t K) {
    srand(10);

    for (uint_fast32_t i = 0; i < N; i += 1) {
        sx[i].x = (float)rand() / RAND_MAX;
        sx[i].y = (float)rand() / RAND_MAX;
        sx[i].id = 0;
    }

    for (uint_fast32_t i = 0; i < K; i += 1) {
        cx[i].x = sx[i].x;
        cx[i].y = sx[i].y;
        cx[i]._c = 0;
        cx[i]._x = 0;
        cx[i]._y = 0;
    }
}

int main(int argc, char** argv) {
    // set the device to use
    hipSetDevice(0);

    uint32_t N = atoi(argv[1]);
    uint32_t K = atoi(argv[2]);


    sample* sx = (sample*)malloc(N * sizeof(sample));
    cluster* cx = (cluster*)malloc(K * sizeof(cluster));

    // Wouldn't it be hillarious if this was the bottleneck?
    // Whether or not it is, it's required by the assignment.
    generate_samples(sx, cx, N, K);

    // get current time
    clock_t start = clock();

    uint32_t iters = k_means_cuda(sx, cx, N, K);

    // elapsed time
    clock_t elapsed = clock() - start;

    // Print the results
    printf("N = %d, K = %d\n", N, K);
    for (uint32_t i = 0; i < K; i++) {
        printf("Center: (%.3f, %.3f) : Size: %d\n",
            cx[i].x, cx[i].y, cx[i]._c);
    }
    printf("Iterations: %d\n", iters);

    // print the time
    printf("Time: %f\n", (float)elapsed / CLOCKS_PER_SEC);

    free(sx);
    free(cx);
}